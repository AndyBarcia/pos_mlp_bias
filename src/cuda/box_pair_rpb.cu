#include "hip/hip_runtime.h"
#include <ATen/native/cuda/KernelUtils.cuh>
#include <c10/util/Half.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>

#include "utils.h"

const int THREADS_PER_BLOCK_FORWARD = 512;
const int THREADS_PER_BLOCK = 256;

template <int C_HIDDEN>
__global__ void box_pair_rbp_kernel(
    const float* __restrict__ mlp_weights,
    const float* __restrict__ boxes1,
    const float* __restrict__ boxes2,
    int B,
    int N1,
    int N2,
    float* __restrict__ output
) {
    // Calculate the total number of output elements (box pairs)
    const int total_elements = B * N1 * N2;
    // Get the global thread ID
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check to ensure the thread is within the problem size
    if (tid >= total_elements) return;

    // Map the 1D thread ID to 3D indices (batch, box1_index, box2_index)
    const int b = tid / (N1 * N2);
    const int n1n2_idx = tid % (N1 * N2);
    const int n1 = n1n2_idx / N2;
    const int n2 = n1n2_idx % N2;

    // Pointers to the specific box data for this thread
    const float* p_box1 = boxes1 + (b * N1 + n1) * 4;
    const float* p_box2 = boxes2 + (b * N2 + n2) * 4;

    // Load box parameters
    const float cx1 = p_box1[0];
    const float cy1 = p_box1[1];
    const float w1 = p_box1[2];
    const float h1 = p_box1[3];

    const float cx2 = p_box2[0];
    const float cy2 = p_box2[1];
    const float w2 = p_box2[2];
    const float h2 = p_box2[3];

    // Compute relative features between the pair of boxes
    const float epsilon = 1e-6f;
    const float dx = (cx2 - cx1) / (w1 + epsilon);
    const float dy = (cy2 - cy1) / (h1 + epsilon);
    const float dw = logf(w2 / (w1 + epsilon));
    const float dh = logf(h2 / (h1 + epsilon));

    // Pointers to the different sections of the MLP weights
    const float* w1_ptr = mlp_weights;
    const float* b1_ptr = mlp_weights + 4 * C_HIDDEN;
    const float* w2_ptr = mlp_weights + 5 * C_HIDDEN;
    const float b2_val = mlp_weights[6 * C_HIDDEN];

    // Temporary array to hold the hidden layer activations
    float temp[C_HIDDEN];
    float max_val = -FLT_MAX;

    // First MLP layer
    for (int k = 0; k < C_HIDDEN; k++) {
        const float w_dx = w1_ptr[k];
        const float w_dy = w1_ptr[C_HIDDEN + k];
        const float w_dw = w1_ptr[2 * C_HIDDEN + k];
        const float w_dh = w1_ptr[3 * C_HIDDEN + k];
        const float b1 = b1_ptr[k];
        
        temp[k] = dx * w_dx + dy * w_dy + dw * w_dw + dh * w_dh + b1;
        if (temp[k] > max_val) max_val = temp[k];
    }

    // Softmax activation function (numerically stable implementation)
    float sum_exp = 0.0f;
    for (int k = 0; k < C_HIDDEN; k++) {
        temp[k] = __expf(temp[k] - max_val);
        sum_exp += temp[k];
    }

    const float inv_sum_exp = 1.0f / sum_exp;
    for (int k = 0; k < C_HIDDEN; k++) {
        temp[k] *= inv_sum_exp;
    }

    // Second MLP layer
    float out_val = 0.0f;
    for (int k = 0; k < C_HIDDEN; k++) {
        out_val += temp[k] * w2_ptr[k];
    }
    out_val += b2_val;

    // Write the final result to the output tensor
    output[tid] = out_val;
}

template <int MAX_C_HIDDEN>
__global__ void box_pair_rbp_dynamic_kernel(
    const float* __restrict__ mlp_weights,
    const float* __restrict__ boxes1,
    const float* __restrict__ boxes2,
    int B,
    int N1,
    int N2,
    int C_HIDDEN,
    float* __restrict__ output
) {
    // Calculate the total number of output elements (box pairs)
    const int total_elements = B * N1 * N2;
    // Get the global thread ID
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check to ensure the thread is within the problem size
    if (tid >= total_elements) return;

    // Map the 1D thread ID to 3D indices (batch, box1_index, box2_index)
    const int b = tid / (N1 * N2);
    const int n1n2_idx = tid % (N1 * N2);
    const int n1 = n1n2_idx / N2;
    const int n2 = n1n2_idx % N2;

    // Pointers to the specific box data for this thread
    const float* p_box1 = boxes1 + (b * N1 + n1) * 4;
    const float* p_box2 = boxes2 + (b * N2 + n2) * 4;

    // Load box parameters
    const float cx1 = p_box1[0];
    const float cy1 = p_box1[1];
    const float w1 = p_box1[2];
    const float h1 = p_box1[3];

    const float cx2 = p_box2[0];
    const float cy2 = p_box2[1];
    const float w2 = p_box2[2];
    const float h2 = p_box2[3];

    // Compute relative features between the pair of boxes
    const float epsilon = 1e-6f;
    const float dx = (cx2 - cx1) / (w1 + epsilon);
    const float dy = (cy2 - cy1) / (h1 + epsilon);
    const float dw = logf(w2 / (w1 + epsilon));
    const float dh = logf(h2 / (h1 + epsilon));

    // Pointers to the different sections of the MLP weights
    const float* w1_ptr = mlp_weights;
    const float* b1_ptr = mlp_weights + 4 * C_HIDDEN;
    const float* w2_ptr = mlp_weights + 5 * C_HIDDEN;
    const float b2_val = mlp_weights[6 * C_HIDDEN];

    // Temporary array to hold the hidden layer activations
    float temp[MAX_C_HIDDEN];
    float max_val = -FLT_MAX;

    // First MLP layer
    for (int k = 0; k < C_HIDDEN; k++) {
        const float w_dx = w1_ptr[k];
        const float w_dy = w1_ptr[C_HIDDEN + k];
        const float w_dw = w1_ptr[2 * C_HIDDEN + k];
        const float w_dh = w1_ptr[3 * C_HIDDEN + k];
        const float b1 = b1_ptr[k];
        
        temp[k] = dx * w_dx + dy * w_dy + dw * w_dw + dh * w_dh + b1;
        if (temp[k] > max_val) max_val = temp[k];
    }

    // Softmax activation function (numerically stable implementation)
    float sum_exp = 0.0f;
    for (int k = 0; k < C_HIDDEN; k++) {
        temp[k] = __expf(temp[k] - max_val);
        sum_exp += temp[k];
    }

    const float inv_sum_exp = 1.0f / sum_exp;
    for (int k = 0; k < C_HIDDEN; k++) {
        temp[k] *= inv_sum_exp;
    }

    // Second MLP layer
    float out_val = 0.0f;
    for (int k = 0; k < C_HIDDEN; k++) {
        out_val += temp[k] * w2_ptr[k];
    }
    out_val += b2_val;

    // Write the final result to the output tensor
    output[tid] = out_val;
}

torch::Tensor fused_box_pair_rpb_forward(
    const torch::Tensor& mlp_weights, // ([4*C' + C' + 1*C' + 1])
    const torch::Tensor& pos1,    // (B,N1,[x,y,w,h])
    const torch::Tensor& pos2,    // (B,N2,[x,y,w,h])
    const int c_hidden
) {
    CHECK_INPUT(mlp_weights);
    CHECK_INPUT(pos1);
    CHECK_INPUT(pos2);
    TORCH_CHECK(c_hidden <= 16, "c_hidden must be <= 16");

    const int B = pos1.size(0);
    const int N1 = pos1.size(1);
    const int N2 = pos2.size(1);

    auto output = torch::empty({B, N1, N2}, mlp_weights.options());
    const int total_elements = B * N1 * N2;
    const int blocks = (total_elements + THREADS_PER_BLOCK_FORWARD - 1) / THREADS_PER_BLOCK_FORWARD;

    if (c_hidden == 16) {
        box_pair_rbp_kernel<16><<<blocks, THREADS_PER_BLOCK_FORWARD>>>(
            mlp_weights.data_ptr<float>(),
            pos1.data_ptr<float>(),
            pos2.data_ptr<float>(),
            B,
            N1,
            N2,
            output.data_ptr<float>()
        );
    } else {
        box_pair_rbp_dynamic_kernel<16><<<blocks, THREADS_PER_BLOCK_FORWARD>>>(
            mlp_weights.data_ptr<float>(),
            pos1.data_ptr<float>(),
            pos2.data_ptr<float>(),
            B,
            N1,
            N2,
            c_hidden,
            output.data_ptr<float>()
        );
    }

    CHECK_CUDA_ERROR(hipPeekAtLastError());

    return output;
}

template <int C_HIDDEN>
__global__ void box_pair_rbp_backward_kernel(
    const float* __restrict__ mlp_weights,
    const float* __restrict__ boxes1,
    const float* __restrict__ boxes2,
    const float* __restrict__ grad_output,
    int B,
    int N1,
    int N2,
    float* __restrict__ grad_weights
) {
    // Each block processes one batch element
    const int b = blockIdx.x;
    const int tid = threadIdx.x;

    extern __shared__ float s_mem[];
    const int grad_size = 6 * C_HIDDEN + 1;

    // Use shared memory for weights and for gradient reduction
    float* s_weights = s_mem;
    float* s_grad = s_mem + grad_size;

    // Use private, register-based array for local gradient accumulation.
    float p_grad[6 * C_HIDDEN + 1];
    for (int i = 0; i < grad_size; ++i) {
        p_grad[i] = 0.0f;
    }

    // Cooperatively load the shared mlp_weights into shared memory.
    if (tid < grad_size) {
        s_weights[tid] = mlp_weights[tid];
    }

    // 0-initialize shared memory for gradients
    if (tid < grad_size) {
        s_grad[tid] = 0.0f;
    }
    __syncthreads();

    // Loop over all pairs of boxes for the current batch element.
    // Each thread computes gradients for a subset of the pairs.
    for (int idx = tid; idx < N1 * N2; idx += blockDim.x) {
        const int n1 = idx / N2;
        const int n2 = idx % N2;

        const float* p_box1 = boxes1 + (b * N1 + n1) * 4;
        const float* p_box2 = boxes2 + (b * N2 + n2) * 4;

        const float cx1 = p_box1[0];
        const float cy1 = p_box1[1];
        const float w1 = p_box1[2];
        const float h1 = p_box1[3];

        const float cx2 = p_box2[0];
        const float cy2 = p_box2[1];
        const float w2 = p_box2[2];
        const float h2 = p_box2[3];

        const float epsilon = 1e-6f;
        const float dx = (cx2 - cx1) / (w1 + epsilon);
        const float dy = (cy2 - cy1) / (h1 + epsilon);
        const float dw = logf(w2 / (w1 + epsilon));
        const float dh = logf(h2 / (h1 + epsilon));

        const float rel_features[4] = {dx, dy, dw, dh};
        
        const float* w1_ptr = s_weights;
        const float* b1_ptr = s_weights + 4 * C_HIDDEN;
        const float* w2_ptr = s_weights + 5 * C_HIDDEN;

        float hidden_activations[C_HIDDEN];
        float softmax_output[C_HIDDEN];
        float max_val = -FLT_MAX;

        for (int k = 0; k < C_HIDDEN; k++) {
            hidden_activations[k] = rel_features[0] * w1_ptr[k]
                                + rel_features[1] * w1_ptr[C_HIDDEN + k]
                                + rel_features[2] * w1_ptr[2 * C_HIDDEN + k]
                                + rel_features[3] * w1_ptr[3 * C_HIDDEN + k]
                                + b1_ptr[k];
            if (hidden_activations[k] > max_val) {
                max_val = hidden_activations[k];
            }
        }
        
        float sum_exp = 0.0f;
        for (int k = 0; k < C_HIDDEN; k++) {
            softmax_output[k] = __expf(hidden_activations[k] - max_val);
            sum_exp += softmax_output[k];
        }

        const float inv_sum_exp = 1.0f / sum_exp;
        for (int k = 0; k < C_HIDDEN; k++) {
            softmax_output[k] *= inv_sum_exp;
        }

        const float dL_doutput = grad_output[b * N1 * N2 + idx];

        // Gradient for b2
        p_grad[6 * C_HIDDEN] += dL_doutput;

        float output_minus_b2 = 0.0f;
        for(int k=0; k<C_HIDDEN; ++k){
            output_minus_b2 += softmax_output[k] * w2_ptr[k];
        }

        // Gradients for w2, w1, and b1
        for (int k = 0; k < C_HIDDEN; k++) {
            // Gradient for w2
            p_grad[5 * C_HIDDEN + k] += dL_doutput * softmax_output[k];

            // Common part of the gradient for the hidden layer
            float dL_dx_k = softmax_output[k] * (dL_doutput * w2_ptr[k] - dL_doutput * output_minus_b2);
            //float dL_dx_k = dL_doutput * softmax_output[k] * (w2_ptr[k] - output_minus_b2);

            // Gradients for w1
            p_grad[k] += dL_dx_k * rel_features[0];                // d/dw1_k0
            p_grad[C_HIDDEN + k] += dL_dx_k * rel_features[1];     // d/dw1_k1
            p_grad[2 * C_HIDDEN + k] += dL_dx_k * rel_features[2]; // d/dw1_k2
            p_grad[3 * C_HIDDEN + k] += dL_dx_k * rel_features[3]; // d/dw1_k3
            
            // Gradient for b1
            p_grad[4 * C_HIDDEN + k] += dL_dx_k;
        }
    }

    __syncthreads();

    // Reduce private gradients (p_grad) into shared memory (s_grad) using warp-level primitives.
    for (int i = 0; i < grad_size; ++i) {
        float val = p_grad[i];
        // Warp-level reduction
        for (int offset = 16; offset > 0; offset /= 2) {
            val += __shfl_down_sync(0xFFFFFFFF, val, offset);
        }
        // First thread in each warp writes the result to shared memory
        if ((tid % 32) == 0) {
            atomicAdd(&s_grad[i], val);
        }
    }
    
    __syncthreads();

    // Atomically add the reduced gradients from this block to the global output tensor.
    if (tid < grad_size) {
        atomicAdd(&grad_weights[tid], s_grad[tid]);
    }
}

torch::Tensor fused_box_pair_rpb_backward(
    const torch::Tensor& grad_out, // (B,N1,N2)
    const torch::Tensor& mlp_weights, // ([4*C' + C' + 1*C' + 1])
    const torch::Tensor& pos1,    // (B,N1,[x,y,w,h])
    const torch::Tensor& pos2,    // (B,N2,[x,y,w,h])
    const int c_hidden
) {
    CHECK_INPUT(mlp_weights);
    CHECK_INPUT(pos1);
    CHECK_INPUT(pos2);
    TORCH_CHECK(c_hidden != 16, "c_hidden must be 16 for the backward pass");

    const int B = pos1.size(0);
    const int N1 = pos1.size(1);
    const int N2 = pos2.size(1);

    auto grad_weights = torch::zeros_like(mlp_weights);
    int grad_size = 6 * c_hidden + 1;
    const size_t shared_mem_size = 2 * grad_size * sizeof(float);

    box_pair_rbp_backward_kernel<16><<<B, THREADS_PER_BLOCK, shared_mem_size>>>(
        mlp_weights.data_ptr<float>(),
        pos1.data_ptr<float>(),
        pos2.data_ptr<float>(),
        grad_out.data_ptr<float>(),
        B,
        N1,
        N2,
        grad_weights.data_ptr<float>()
    );

    CHECK_CUDA_ERROR(hipPeekAtLastError());

    return grad_weights;
}