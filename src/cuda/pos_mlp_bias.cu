#include "hip/hip_runtime.h"
#include <ATen/native/cuda/KernelUtils.cuh>
#include <c10/util/Half.h>
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

#define CHECK_CUDA_ERROR(err) do { cuda_check((err), __FILE__, __LINE__); } while(false)
inline void cuda_check(hipError_t error_code, const char *file, int line)
{
    if (error_code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error %d: %s. In file '%s' on line %d\n", error_code, hipGetErrorString(error_code), file, line);
        fflush(stderr);
        exit(error_code);
    }
}

const int MAX_C = 16;
const int THREADS_PER_BLOCK_FORWARD = 512;
const int THREADS_PER_BLOCK = 256;

__global__ void __launch_bounds__(THREADS_PER_BLOCK_FORWARD, 2) pos_mlp_bias_forward_kernel(
    const float* __restrict__ mlp_weights,
    const float* __restrict__ pos,
    int c_hidden,
    int W,
    int H,
    int B,
    float* __restrict__ output
) {
    const int total_elements = B * H * W;
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= total_elements) return;

    const int b = tid / (H * W);
    const int spatial_idx = tid % (H * W);

    const float i = ((float) (spatial_idx / W)) / (float) (H-1);
    const float j = ((float) (spatial_idx % W)) / (float) (W-1);

    const float cx = pos[b * 4 + 0];
    const float cy = pos[b * 4 + 1];
    const float half_w = fmaxf(pos[b * 4 + 2] * 0.5f, 1e-6f);
    const float half_h = fmaxf(pos[b * 4 + 3] * 0.5f, 1e-6f);

    const float rel_x = (j - cx) / half_w;
    const float rel_y = (i - cy) / half_h;

    const int weights_offset = b * (4 * c_hidden + 1);
    const float* w = &mlp_weights[weights_offset];

    float temp[MAX_C];
    float max_val = -FLT_MAX;
    for (int k = 0; k < c_hidden; k++) {
        const float w_x = w[2 * k];
        const float w_y = w[2 * k + 1];
        const float b1 = w[2 * c_hidden + k];
        temp[k] = rel_x * w_x + rel_y * w_y + b1;
        if (temp[k] > max_val) max_val = temp[k];
    }

    float sum_exp = 0.0f;
    for (int k = 0; k < c_hidden; k++) {
        temp[k] = __expf(temp[k] - max_val);
        sum_exp += temp[k];
    }

    const float inv_sum_exp = 1.0f / sum_exp;
    for (int k = 0; k < c_hidden; k++) {
        temp[k] *= inv_sum_exp;
    }

    float out_val = 0.0f;
    for (int k = 0; k < c_hidden; k++) {
        out_val += temp[k] * w[3 * c_hidden + k];
    }
    out_val += w[4 * c_hidden];

    output[tid] = out_val;
}

torch::Tensor fused_attn_forward(
    const torch::Tensor& mlp_weights, // (B, [2*C' + C' + 1*C' + 1])
    const torch::Tensor& pos,    // (B,[x,y,w,h])
    const int c_hidden,
    const int H,
    const int W
) {
    CHECK_INPUT(mlp_weights);
    CHECK_INPUT(pos);

    const int B = mlp_weights.size(0);

    auto output = torch::empty({B, H, W}, mlp_weights.options());
    const int total_elements = B * H * W;
    const int blocks = (total_elements + THREADS_PER_BLOCK_FORWARD - 1) / THREADS_PER_BLOCK_FORWARD;

    pos_mlp_bias_forward_kernel<<<blocks, THREADS_PER_BLOCK_FORWARD>>>(
        mlp_weights.data_ptr<float>(),
        pos.data_ptr<float>(),
        c_hidden,
        W,
        H,
        B,
        output.data_ptr<float>()
    );

    CHECK_CUDA_ERROR(hipPeekAtLastError());

    return output;
}


__global__ void __launch_bounds__(THREADS_PER_BLOCK, 2) pos_mlp_bias_backward_kernel(
    const float* __restrict__ mlp_weights,
    const float* __restrict__ pos,
    const float* __restrict__ grad_output,
    int c_hidden,
    int W,
    int H,
    int B,
    float* __restrict__ grad_weights
) {

    int b = blockIdx.x;
    int tid = threadIdx.x;

    extern __shared__ float s_grad[];
    int grad_size = 4 * c_hidden + 1;

    if (tid < grad_size) {
        s_grad[tid] = 0.0f;
    }
    __syncthreads();

    const float cx = pos[b * 4 + 0];
    const float cy = pos[b * 4 + 1];
    const float half_w = fmaxf(pos[b * 4 + 2] * 0.5f, 1e-6f);
    const float half_h = fmaxf(pos[b * 4 + 3] * 0.5f, 1e-6f);

    const float* w = &mlp_weights[b * grad_size];
    const float* grad_out_b = &grad_output[b * H * W];

    for (int idx = tid; idx < H * W; idx += blockDim.x) {
        int i = idx / W;
        int j = idx % W;

        float rel_x = ((((float)j)/(float)(W-1)) - cx) / half_w;
        float rel_y = ((((float)i)/(float)(H-1)) - cy) / half_h;

        float x[MAX_C];
        float max_x = -FLT_MAX;
        for (int k = 0; k < c_hidden; k++) {
            x[k] = rel_x * w[2 * k] + rel_y * w[2 * k + 1] + w[2 * c_hidden + k];
            if (x[k] > max_x) max_x = x[k];
        }

        float s[MAX_C];
        float sum_exp = 0.0f;
        for (int k = 0; k < c_hidden; k++) {
            s[k] = __expf(x[k] - max_x);
            sum_exp += s[k];
        }
        float inv_sum = 1.0f / sum_exp;
        for (int k = 0; k < c_hidden; k++) {
            s[k] *= inv_sum;
        }

        float dL_doutput_bij = grad_out_b[i * W + j];

        float output_minus_b2 = 0.0f;
        for (int k = 0; k < c_hidden; k++) {
            output_minus_b2 += s[k] * w[3 * c_hidden + k];
        }

        atomicAdd(&s_grad[4 * c_hidden], dL_doutput_bij);

        for (int k = 0; k < c_hidden; k++) {
            atomicAdd(&s_grad[3 * c_hidden + k], dL_doutput_bij * s[k]);

            float dL_dx_k = s[k] * (dL_doutput_bij * w[3 * c_hidden + k] - dL_doutput_bij * output_minus_b2);

            atomicAdd(&s_grad[2 * k], dL_dx_k * rel_x);
            atomicAdd(&s_grad[2 * k + 1], dL_dx_k * rel_y);
            atomicAdd(&s_grad[2 * c_hidden + k], dL_dx_k);
        }
    }

    __syncthreads();

    if (tid < grad_size) {
        grad_weights[b * grad_size + tid] = s_grad[tid];
    }
}

__global__ void __launch_bounds__(THREADS_PER_BLOCK, 4) pos_mlp_bias_backward_kernel_optimized(
    const float* __restrict__ mlp_weights,
    const float* __restrict__ pos,
    const float* __restrict__ grad_output,
    int c_hidden,
    int W,
    int H,
    int B,
    float* __restrict__ grad_weights
) {

    int b = blockIdx.x;
    int tid = threadIdx.x;

    extern __shared__ float s_grad[];
    int grad_size = 4 * c_hidden + 1;

    // Use a private, register-based array for gradient accumulation.
    // Initialize it to zero.
    float p_grad[4 * MAX_C + 1]; // Use the compile-time constant
    for (int i = 0; i < grad_size; ++i) {
        p_grad[i] = 0.0f;
    }

    // Cooperatively load mlp_weights into shared memory.
    // This access is fully coalesced.
    if (tid < grad_size) {
        s_grad[tid] = mlp_weights[b * grad_size + tid];
    }
    __syncthreads();

    const float cx = pos[b * 4 + 0];
    const float cy = pos[b * 4 + 1];
    const float half_w = fmaxf(pos[b * 4 + 2] * 0.5f, 1e-6f);
    const float half_h = fmaxf(pos[b * 4 + 3] * 0.5f, 1e-6f);

    //const float* w = &mlp_weights[b * grad_size];
    const float* w = s_grad;
    const float* grad_out_b = &grad_output[b * H * W];

    for (int idx = tid; idx < H * W; idx += blockDim.x) {
        int i = idx / W;
        int j = idx % W;

        float rel_x = ((((float)j) / (float)(W - 1)) - cx) / half_w;
        float rel_y = ((((float)i) / (float)(H - 1)) - cy) / half_h;

        float x[MAX_C];
        float s[MAX_C];

        float max_x = -FLT_MAX;
        for (int k = 0; k < c_hidden; k++) {
            x[k] = rel_x * w[2 * k] + rel_y * w[2 * k + 1] + w[2 * c_hidden + k];
            if (x[k] > max_x) max_x = x[k];
        }

        float sum_exp = 0.0f;
        for (int k = 0; k < c_hidden; k++) {
            s[k] = __expf(x[k] - max_x);
            sum_exp += s[k];
        }

        float inv_sum = 1.0f / sum_exp;
        for (int k = 0; k < c_hidden; k++) {
            s[k] *= inv_sum;
        }

        float dL_doutput_bij = grad_out_b[i * W + j];

        float output_minus_b2 = 0.0f;
        for (int k = 0; k < c_hidden; k++) {
            output_minus_b2 += s[k] * w[3 * c_hidden + k];
        }

        p_grad[4 * c_hidden] += dL_doutput_bij;
        for (int k = 0; k < c_hidden; k++) {
            p_grad[3 * c_hidden + k] += dL_doutput_bij * s[k];
            float dL_dx_k = s[k] * (dL_doutput_bij * w[3 * c_hidden + k] - dL_doutput_bij * output_minus_b2);
            p_grad[2 * k] += dL_dx_k * rel_x;
            p_grad[2 * k + 1] += dL_dx_k * rel_y;
            p_grad[2 * c_hidden + k] += dL_dx_k;
        }
    }

    __syncthreads();

    // Re-Initialize shared memory in parallel for reduction.
    if (tid < grad_size) {
        s_grad[tid] = 0.0f;
    }

    __syncthreads();

    // Each thread contributes its p_grad values.
    for (int i = 0; i < grad_size; ++i) {
        float val = p_grad[i];

        // Perform a warp-level reduction using shuffle instructions.
        for (int offset = 16; offset > 0; offset /= 2) {
            val += __shfl_down_sync(0xFFFFFFFF, val, offset);
        }

        // The first thread in each warp (the "warp leader") writes the
        // partial sum for its warp to shared memory. This greatly reduces contention.
        if ((tid % 32) == 0) {
            atomicAdd(&s_grad[i], val);
        }
    }
    
    __syncthreads();

    // Write final result from shared memory to global memory.
    if (tid < grad_size) {
        grad_weights[b * grad_size + tid] = s_grad[tid];
    }
}

torch::Tensor fused_attn_backward(
    const torch::Tensor& grad_out, // (B, H, W)
    const torch::Tensor& mlp_weights, // (B, [2*C' + C' + 1*C' + 1])
    const torch::Tensor& pos,    // (B,[x,y,w,h])
    const int c_hidden
) {
    CHECK_INPUT(grad_out);
    CHECK_INPUT(mlp_weights);
    CHECK_INPUT(pos);

    const int B = mlp_weights.size(0);
    const int H = grad_out.size(1);
    const int W = grad_out.size(2);

    auto grad_weights = torch::zeros_like(mlp_weights);
    int grad_size = 4 * c_hidden + 1;
    size_t shared_mem_size = grad_size * sizeof(float);

    pos_mlp_bias_backward_kernel_optimized<<<B, THREADS_PER_BLOCK, shared_mem_size>>>(
        mlp_weights.data_ptr<float>(),
        pos.data_ptr<float>(),
        grad_out.data_ptr<float>(),
        c_hidden,
        W,
        H,
        B,
        grad_weights.data_ptr<float>()
    );

    CHECK_CUDA_ERROR(hipPeekAtLastError());

    return grad_weights;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("forward", &fused_attn_forward, "Fused Attention Forward (Multi-Head)");
    m.def("backward", &fused_attn_backward, "Fused Attention Backward (Multi-Head)");
}